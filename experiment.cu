#include "hip/hip_runtime.h"
/* vim: set smartindent expandtab shiftwidth=4 softtabstop=4: */

#include <hiprand/hiprand_kernel.h>
#include <cutil.h>
#include <cutil_inline_runtime.h>
#include <vector>
#include <mpi.h>
#include "experiment.h"

typedef size_t Symbol;
typedef size_t State;
typedef int Shift; 

struct Command {
    State newState;
    Symbol newSymbol;
    Shift shift;
};


__device__ __host__ __inline__ size_t getProgramSize(
        const size_t stateCount,
        const size_t alphabetSize) {
    return stateCount * alphabetSize;
}


__host__ Command *allocPrograms(
        const size_t stateCount, 
        const size_t alphabetSize,
        const size_t iterationCount) {
    Command *pPrograms = 0;
    const size_t programSize = getProgramSize(stateCount, alphabetSize);
    cutilSafeCall(hipMalloc((void **)&pPrograms, 
        iterationCount * programSize * sizeof(Command)));
    return pPrograms;
}


__host__ void freePrograms(Command * const pPrograms) {
    cutilSafeCall(hipFree(pPrograms));
}


__inline__ __device__ State getRandomState(
        hiprandState * const pRNGState, 
        const size_t stateCount) {
    return hiprand(pRNGState) % (stateCount + 1);
}


__inline__ __device__ Symbol getRandomSymbol(
        hiprandState * const pRNGState, 
        const size_t alphabetSize) {
    return hiprand(pRNGState) % alphabetSize;
}


__inline__ __device__ Shift getRandomShift(
        hiprandState * const pRNGState, 
        const bool stopActionUsed) {
    if (stopActionUsed) {
        return hiprand(pRNGState) % 3 - 1;
    } else {
        return 2 * (hiprand(pRNGState) % 2) - 1;
    }
}


__inline__ __device__ void createRandomCommand(
        hiprandState * const pRNGState, 
        const size_t stateCount, 
        const bool stopActionUsed,
        const size_t alphabetSize,
        Command * const pCommand) {
    pCommand->newState = getRandomState(pRNGState, stateCount);
    pCommand->newSymbol = getRandomSymbol(pRNGState, alphabetSize);
    pCommand->shift = getRandomShift(pRNGState, stopActionUsed);
}


__global__ void createRandomPrograms(
        const size_t stateCount, 
        const bool stopActionUsed, 
        const size_t alphabetSize,
        hiprandState * const pRNGStates, 
        Command * const pPrograms) {
    const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState * const pRNGState = pRNGStates + index;
    const size_t programSize = getProgramSize(stateCount, alphabetSize);
    Command * const pProgram = pPrograms + programSize * index;
    for (size_t i = 0; i < programSize; ++i) {
        createRandomCommand(pRNGState, stateCount, stopActionUsed, 
            alphabetSize, pProgram + i);
    }
}


__host__ hiprandState *allocRNGStates(const size_t iterationCount) {
    hiprandState *pRNGStates = 0;
    cutilSafeCall(hipMalloc((void **)&pRNGStates, 
        iterationCount * sizeof(hiprandState)));
    return pRNGStates;
}


__host__ void freeRNGStates(hiprandState * const pRNGStates) {
    cutilSafeCall(hipFree(pRNGStates));
}


__global__ void createRNGStates(
        const size_t seed, 
        hiprandState * const pRNGStates) {
    const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, index, 0, pRNGStates + index);
}


__device__ __host__ __inline__ size_t getTapeSize(
        const size_t toMaxStepCount) {
    return 2 * toMaxStepCount + 1;
}


__host__ Symbol *allocTapes(
        const size_t toMaxStepCount, 
        const size_t iterationCount) {
    Symbol *pTapes = 0;
    const size_t tapeSize = getTapeSize(toMaxStepCount);
    cutilSafeCall(hipMalloc((void **)&pTapes, 
        tapeSize * iterationCount * sizeof(Symbol)));
    return pTapes;
}


__host__ void freeTapes(Symbol * const pTapes) {
    cutilSafeCall(hipFree(pTapes));
}


__host__ void initTapes(Symbol * const pTapes, 
        const size_t iterationCount, 
        const size_t toMaxStepCount) {
    const size_t tapeSize = getTapeSize(toMaxStepCount);
    cutilSafeCall(hipMemset(pTapes, 0, 
        tapeSize * iterationCount * sizeof(Symbol)));
}


__host__ State *allocStates(const size_t iterationCount) {
    State *pStates = 0;
    cutilSafeCall(hipMalloc((void **)&pStates, 
        iterationCount * sizeof(State)));
    return pStates;
}


__host__ void freeStates(State * const pStates) { 
    cutilSafeCall(hipFree(pStates));
}


__host__ void initStates(
        State * const pStates, 
        const size_t iterationCount) {
    cutilSafeCall(hipMemset(pStates, 0, iterationCount * sizeof(State)));
}


__host__ size_t *allocPositions(const size_t iterationCount) {
    size_t *pPositions = 0;
    cutilSafeCall(hipMalloc((void **)&pPositions, 
        iterationCount * sizeof(size_t)));
    return pPositions;
}


__host__ void freePositions(size_t * const pPositions) {
    cutilSafeCall(hipFree(pPositions));
}


__global__ void initPositions(
        size_t * const pPositions, 
        const size_t toMaxStepCount) {
    const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    pPositions[index] = toMaxStepCount;
}


__host__ size_t getMachineInStateCount(
        const size_t iterationCount, 
        const State * const pDeviceStates,
        const State state) {
    size_t * const pHostStates = new State[iterationCount];
    cutilSafeCall(hipMemcpy(pHostStates, pDeviceStates, 
        iterationCount * sizeof(State), hipMemcpyDeviceToHost));
    size_t machineInStateCount = 0;
    for (size_t i = 0; i < iterationCount; ++i) {
        if (pHostStates[i] == state) {
            machineInStateCount += 1;
        }
    }
    delete [] pHostStates;
    return machineInStateCount;
}


__global__ void runMachines(
        const size_t stateCount,
        const size_t toMaxStepCount,
        const size_t alphabetSize,
        const Command * const pPrograms,
        const size_t stepCount,
        const State finalState,
        Symbol * const pTapes,
        State * const pStates,
        size_t * const pPositions) {
    const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t programSize = getProgramSize(stateCount, alphabetSize);
    const size_t tapeSize = getTapeSize(toMaxStepCount);
    const Command * const pProgram = pPrograms + programSize * index;
    Symbol * const pTape = pTapes + tapeSize * index;
    State state = pStates[index];
    size_t position = pPositions[index];
    for (size_t i = 0; i < stepCount; ++i) {
        if (state == finalState) {
            break;	
        }
        Symbol * const pSymbol = pTape + position;
        const Command * const pCommand = 
            pProgram + alphabetSize * state + *pSymbol;
        *pSymbol = pCommand->newSymbol;
        position += pCommand->shift;
        state = pCommand->newState;		
    }
    pStates[index] = state;
    pPositions[index] = position;
}


__global__ void runMachinesOnHalfTape(
        const size_t stateCount,
        const size_t toMaxStepCount,
        const size_t alphabetSize,
        const Command * const pPrograms,
        const size_t stepCount,
        const State finalState,
        const State brokenState,
        Symbol * const pTapes,
        State * const pStates,
        size_t * const pPositions) {
    const size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    const size_t programSize = getProgramSize(stateCount, alphabetSize);
    const size_t tapeSize = getTapeSize(toMaxStepCount);
    const Command * const pProgram = pPrograms + programSize * index;
    Symbol * const pTape = pTapes + tapeSize * index;
    State state = pStates[index];
    size_t position = pPositions[index];
    for (size_t i = 0; i < stepCount; ++i) {
        if (state == finalState || state == brokenState) {
            break;	
        }
        Symbol * const pSymbol = pTape + position;
        const Command * const pCommand = 
            pProgram + alphabetSize * state + *pSymbol;
        *pSymbol = pCommand->newSymbol;
        position += pCommand->shift;
        if (position < toMaxStepCount) {
            state = brokenState;
            break;
        }
        state = pCommand->newState;		
    }
    pStates[index] = state;
    pPositions[index] = position;
}


__host__ size_t getToMaxStepCount(const ExperimentParameters &ep) {
    size_t toMaxStepCount = ep.fromMaxStepCount;
    for (size_t i = 0; i <= ep.countMaxStepCount; ++i) {
            toMaxStepCount *= ep.mDeltaMaxStepCount;
    }
    return toMaxStepCount;
}


__host__ size_t getToStateCount(const ExperimentParameters &ep) {
    size_t toStateCount = ep.fromStateCount;
    for (size_t i = 0; i <= ep.countStateCount; ++i) {
            toStateCount += ep.deltaStateCount;
    }
    return toStateCount;
}


__host__ void runExperiment(
        const ExperimentParameters &ep,
        const size_t stateCount, 
        hiprandState * const pRNGStates,
        Command * const pPrograms,
        Symbol * const pTapes,
        State * const pStates,
        size_t * const pPositions,
        size_t * const finishedMachineCounts,
        size_t * const brokenMachineCounts) { 
    initStates(pStates, ep.iterationCount);
    const dim3 threads = dim3(ep.threadBlockSize, 1, 1);
    const dim3 blocks = dim3(ep.iterationCount / threads.x, 1, 1);
    initPositions<<<blocks, threads>>>(pPositions, getToMaxStepCount(ep));
    cutilSafeCall(hipDeviceSynchronize());
    initTapes(pTapes, ep.iterationCount, getToMaxStepCount(ep));
    createRandomPrograms<<<blocks, threads>>>(stateCount, ep.stopActionUsed, 
        ep.alphabetSize, pRNGStates, pPrograms);
    cutilSafeCall(hipDeviceSynchronize());
    const State finalState = stateCount;
    const State brokenState = stateCount + 1;
    size_t from = 0;
    size_t to = ep.fromMaxStepCount;
    for (size_t j = 0; j <= ep.countMaxStepCount; ++j) {
        if (!ep.halfTapeUsed) {
            runMachines<<<blocks, threads>>>(stateCount, getToMaxStepCount(ep),
                ep.alphabetSize, pPrograms, to - from, finalState, pTapes, 
                pStates, pPositions);
            cutilSafeCall(hipDeviceSynchronize());
            finishedMachineCounts[j] += 
                getMachineInStateCount(ep.iterationCount, pStates, finalState);
        } else {
            runMachinesOnHalfTape<<<blocks, threads>>>(stateCount, 
                getToMaxStepCount(ep), ep.alphabetSize, 
                pPrograms, to - from, finalState, brokenState, pTapes,
                pStates, pPositions);
            cutilSafeCall(hipDeviceSynchronize());
            finishedMachineCounts[j] +=
                getMachineInStateCount(ep.iterationCount, pStates, finalState);
            brokenMachineCounts[j] += 
                getMachineInStateCount(ep.iterationCount, pStates, brokenState);
        }
        from = to;
        to = to * ep.mDeltaMaxStepCount;
    }
}

__host__ void runExperiment (
        const ExperimentParameters &ep, 
        const size_t threadCount,
        const size_t threadId,
        size_t *finishedMachineCounts,
        size_t *brokenMachineCounts) {
    hiprandState * const pRNGStates = allocRNGStates(ep.iterationCount);
    const dim3 threads = dim3(ep.threadBlockSize, 1, 1);
    const dim3 blocks = dim3(ep.iterationCount / threads.x, 1, 1);
    createRNGStates<<<blocks, threads>>>(ep.seed, pRNGStates);
    cutilSafeCall(hipDeviceSynchronize());
    Command * const pPrograms = allocPrograms(getToStateCount(ep), 
        ep.alphabetSize, ep.iterationCount);
    Symbol * const pTapes = allocTapes(getToMaxStepCount(ep), 
        ep.iterationCount);
    State * const pStates = allocStates(ep.iterationCount);
    size_t * const pPositions = allocPositions(ep.iterationCount);
    for (size_t i = 0; i <= ep.countStateCount; ++i) { 
        if (i % threadCount == threadId) {
            for (size_t run = 0; run < ep.runCount; ++run) {
                runExperiment(ep, ep.fromStateCount + i * ep.deltaStateCount,
                        pRNGStates, pPrograms, pTapes, pStates, pPositions, 
                        finishedMachineCounts + i * (ep.countMaxStepCount + 1), 
                        brokenMachineCounts + i * (ep.countMaxStepCount + 1));
            }
        }
    }
    freePositions(pPositions);
    freeStates(pStates);
    freeTapes(pTapes);
    freePrograms(pPrograms);
    freeRNGStates(pRNGStates);
}


size_t getResultSize(const ExperimentParameters &ep) {
    return (ep.countStateCount + 1) * (ep.countMaxStepCount + 1);
}


void runExperiment(
        const ExperimentParameters &ep, 
        std::vector<size_t> &finishedMachineCounts, 
        std::vector<size_t> &brokenMachineCounts) {
    int threadId;
    MPI_Comm_rank(MPI_COMM_WORLD, &threadId);
    int threadCount;
    MPI_Comm_size(MPI_COMM_WORLD, &threadCount);
    size_t localThreadId = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
    size_t deviceId = localThreadId;
    cutilSafeCall(hipSetDevice(deviceId));
    if (threadId == 0) {
        std::cout << "# threadCount=" << threadCount << std::endl;
    }
    std::cout << "# thread" << threadId << ".deviceId=" << deviceId << std::endl;
    std::cout << "# thread" << threadId << ".localThreadId=" << localThreadId << std::endl;
    hipDeviceProp_t deviceProperties;
    cutilSafeCall(hipGetDeviceProperties(&deviceProperties, deviceId));
    std::cout << "# thread" << threadId << ".deviceName=" << deviceProperties.name << std::endl;
    char nodeName[256];
    gethostname(nodeName, sizeof(nodeName));
    std::cout << "# thread" << threadId << ".nodeName=" << nodeName << std::endl;

    size_t resultSize = getResultSize(ep);
    runExperiment(ep, threadCount, threadId, &finishedMachineCounts[0], 
        &brokenMachineCounts[0]);
    size_t *allFinishedMachineCounts;
    size_t *allBrokenMachineCounts;
    if (threadId == 0) {
        allFinishedMachineCounts = new size_t[resultSize * threadCount];
        allBrokenMachineCounts = new size_t[resultSize * threadCount];
        
    }
    MPI_Datatype MPI_SIZE_T;
    if (sizeof(size_t) == sizeof(unsigned long)) {
        MPI_SIZE_T = MPI_UNSIGNED_LONG;
    } else if (sizeof(size_t) == sizeof(unsigned int)) {
        MPI_SIZE_T = MPI_UNSIGNED;
    } else {
        std::cerr << "The type size_t has unknown size" << std::endl;
        exit(EXIT_FAILURE);
    }
    MPI_Gather(&finishedMachineCounts[0], resultSize, MPI_SIZE_T,
            allFinishedMachineCounts, resultSize, MPI_SIZE_T, 
            0, MPI_COMM_WORLD);
    MPI_Gather(&brokenMachineCounts[0], resultSize, MPI_SIZE_T,
            allBrokenMachineCounts, resultSize, MPI_SIZE_T, 
            0, MPI_COMM_WORLD);
    if (threadId == 0) {
        for (size_t i = 1; i < threadCount; ++i) {
            for (size_t j = 0; j < resultSize; ++j) {
                size_t index = j + i *resultSize;
                finishedMachineCounts[j] += allFinishedMachineCounts[index];
                brokenMachineCounts[j] += allBrokenMachineCounts[index];
            }
        }
        delete [] allFinishedMachineCounts;
        delete [] allBrokenMachineCounts;
    }
}

